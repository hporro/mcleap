#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <glm/glm.hpp>
#include <random>

#include "Host_Triangulation.h"
#include "Helpers_Triangulation.h"
#include "Device_Triangulation.h"

#include <tinytest.h>

template<int num_vertices, int max_ring_neighbors>
void test_closest_neighbors(double movement, double bounds) {
	int numP = num_vertices;

	std::vector<glm::vec2> h_pos, h_move;
	glm::vec2* d_move;
	hipMalloc((void**)&d_move, numP * sizeof(glm::vec2));
	//std::random_device dev;
	std::mt19937 rng{ 3 };

	std::uniform_real_distribution<float> pos_r(-bounds, bounds);
	std::uniform_real_distribution<float> move_r(-movement, movement);

	for (int i = 0; i < numP; i++) {
		h_pos.push_back(glm::vec2(pos_r(rng), pos_r(rng)));
	}
	for (int i = 0; i < numP; i++) {
		h_move.push_back(glm::vec2(move_r(rng), move_r(rng)));
	}
	hipMemcpy(d_move, h_move.data(), numP * sizeof(glm::vec2), hipMemcpyHostToDevice);

	HostTriangulation* ht = new HostTriangulation();
	ht->addDelaunayPoints(h_pos);

	int* d_ring_neighbors, * h_ring_neighbors;
	hipMalloc((void**)&d_ring_neighbors, ht->m_pos.size() * max_ring_neighbors * sizeof(int));

	int* d_closest_neighbors, * h_closest_neighbors = new int[ht->m_pos.size()];
	hipMalloc((void**)&d_closest_neighbors, ht->m_pos.size() * sizeof(int));

	DeviceTriangulation dt(ht);
	//dt.untangle();
	//dt.delonize();
	dt.oneRing<max_ring_neighbors>(d_ring_neighbors);
	dt.closestNeighbors<max_ring_neighbors>(d_ring_neighbors, d_closest_neighbors);

	hipMemcpy(h_closest_neighbors, d_closest_neighbors, ht->m_pos.size() * sizeof(int), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	int* real_closest_neighbors = new int[ht->m_pos.size()];

	for (int i = 0; i < ht->m_pos.size(); i++)real_closest_neighbors[i] = -1;

	for (int i = 0; i < ht->m_pos.size(); i++) {
		glm::vec2 i_pos = ht->m_pos[i];
		float closest_dist = sqrtDist(i_pos, ht->m_pos[0]); if (i == 0)closest_dist = sqrtDist(i_pos, ht->m_pos[1]);
		int closest_neighbor = 0; if (i == 0)closest_neighbor = 1;

		for (int j = 0; j < ht->m_pos.size(); j++) {
			if (i == j)continue;
			float curr_dist = sqrtDist(i_pos, ht->m_pos[j]);
			if (curr_dist < closest_dist) {
				closest_dist = curr_dist;
				closest_neighbor = j;
			}
		}
		real_closest_neighbors[i] = closest_neighbor;
	}


	for (int i = 0; i < ht->m_pos.size(); i++) {
		//printf("i: %d real: %d computed: %d\n", i, real_closest_neighbors[i], h_closest_neighbors[i]);
		ASSERT_EQUALS(real_closest_neighbors[i], h_closest_neighbors[i]);
	}

	delete ht;
	delete[] h_closest_neighbors;
	delete[] real_closest_neighbors;
	hipFree(d_move);
	hipFree(d_ring_neighbors);
	hipFree(d_closest_neighbors);
}

int main(int argc, char* argv[]) {
	RUN((test_closest_neighbors<10, 100>), 0.1, 1000.0);
	RUN((test_closest_neighbors<100, 100>), 0.1, 1000.0);
	RUN((test_closest_neighbors<1000, 100>), 0.1, 1000.0);
	return TEST_REPORT();
}