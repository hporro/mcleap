#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <glm/gtx/matrix_transform_2d.hpp>
#include <glm/vec2.hpp>
#include <random>

#include <math.h>

#include "predicates.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include <stb_image_write.h>

#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

constexpr double H_PI = 3.14159265358979323846;

template <class T>
__device__ __host__ inline T pow2(const T &a)
{
    return a * a;
}

inline __device__ __host__ double cross(const glm::dvec2 &a, const glm::dvec2 &b)
{
    return (a.x * b.y) - (a.y * b.x);
}

inline __device__ __host__ double dot(const glm::dvec2 &a, const glm::dvec2 &b)
{
    return (a.x * b.x) + (a.y * b.y);
}

inline __device__ __host__ double d_angle_vectors(const glm::dvec2 &u, const glm::dvec2 &v)
{
    return atan2(abs(cross(u, v)), dot(u, v));
}

// taken from https://gitlab.com/hporro01/mcleap/-/blob/main/src/kernels.cuh
///         com_a
///           +
///          /|\
///         / | \
///        /  |  \
///       /   |   \
///      / \  |  / \
/// op1 + α | | | β + op2
///      \ /  |  \ /
///       \   |   /
///        \  |  /
///         \ | /
///          \|/
///           +
///         com_b
/// Computes wether or not we have to flip (either 0 or 1). It is 1 if α+β>PI+EPS
inline __device__ __host__ double angle_incircle(const glm::dvec2 &com_a, const glm::dvec2 &op1, const glm::dvec2 &com_b, const glm::dvec2 &op2)
{
    glm::dvec2 u;    // vector
    glm::dvec2 p, q; // points
    // get two vectors of the first triangle
    p = op1;
    q = com_a;
    u = q - p; //! + 5 flop
    q = com_b;
    double alpha = d_angle_vectors(u, q - p);
    // the same for other triangle
    p = op2;
    q = com_a;
    u = q - p;
    q = com_b;
    double beta = d_angle_vectors(u, q - p);

    return fabs(alpha + beta) / H_PI;
}

__device__ __host__ inline double matrix_incircle(const glm::dvec2 &a, const glm::dvec2 &b, const glm::dvec2 &c, const glm::dvec2 &d)
{
    const double a00 = a.x - d.x;
    const double a01 = a.y - d.y;
    const double a02 = pow2(a00) + pow2(a01);
    const double a10 = b.x - d.x;
    const double a11 = b.y - d.y;
    const double a12 = pow2(a10) + pow2(a11);
    const double a20 = c.x - d.x;
    const double a21 = c.y - d.y;
    const double a22 = pow2(a20) + pow2(a21);

    double det = (a00 * (a11 * a22 - a12 * a21) - a01 * (a10 * a22 - a12 * a20) + a02 * (a10 * a21 - a11 * a20));

    return det;
    // return (a00 * (a11 * a22 - a12 * a21) - a01 * (a10 * a22 - a12 * a20) + a02 * (a10 * a21 - a11 * a20)) > EPS;
}

__host__ inline double exact_incircle(const glm::dvec2 &a, const glm::dvec2 &b, const glm::dvec2 &c, const glm::dvec2 &d)
{
    double a_arr[2] = {a.x, a.y};
    double b_arr[2] = {b.x, b.y};
    double c_arr[2] = {c.x, c.y};
    double d_arr[2] = {d.x, d.y};

    return incircle(a_arr, b_arr, c_arr, d_arr);
}

#define EPS 1e-10

__global__ void calc_predicate_incircle(int resolution, glm::dvec2 a, glm::dvec2 b, glm::dvec2 c, glm::dvec2 d, double reasonable_eps, double *dest)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = idx % resolution;
    const int i = idx / resolution;
    if (i < resolution)
    {
        glm::dvec2 perturbation((i - resolution / 2) * reasonable_eps, (j - resolution / 2) * reasonable_eps);
        dest[i * resolution + j] = matrix_incircle(a, b, c, d + perturbation);
    }
}

#undef EPS

__global__ void calc_predicate_angle_incircle(int resolution, glm::dvec2 a, glm::dvec2 b, glm::dvec2 c, glm::dvec2 d, double reasonable_eps, double *dest)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = idx % resolution;
    const int i = idx / resolution;
    if (i < resolution)
    {
        glm::dvec2 perturbation((i - resolution / 2) * reasonable_eps, (j - resolution / 2) * reasonable_eps);
        dest[i * resolution + j] = angle_incircle(a, b, c, d + perturbation);
    }
}

__global__ void get_image_from_angle_incircle(int resolution, double *org, char *dest)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = idx % resolution;
    const int i = idx / resolution;
    if (idx < resolution * resolution)
    {
        // printf("idx: %d val: %lf\n", idx, org[(i * resolution + j)]);
        dest[(i + resolution * j) * 3 + 0] = (org[(i * resolution + j)] > 1) * 254;
        dest[(i + resolution * j) * 3 + 1] = (org[(i * resolution + j)] == 1) * 254;
        dest[(i + resolution * j) * 3 + 2] = (org[(i * resolution + j)] < 1) * 254;
    }
}

__global__ void get_image_from_incircle(int resolution, double *org, char *dest)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = idx % resolution;
    const int i = idx / resolution;
    if (idx < resolution * resolution)
    {
        // printf("idx: %d val: %lf\n", idx, org[(i * resolution + j)]);
        dest[(i + resolution * j) * 3 + 0] = (org[(i * resolution + j)] > 0) * 254;
        dest[(i + resolution * j) * 3 + 1] = (org[(i * resolution + j)] == 0) * 254;
        dest[(i + resolution * j) * 3 + 2] = (org[(i * resolution + j)] < 0) * 254;
    }
}

glm::dvec2 rotate2d(const glm::dvec2 &a, double angle)
{
    double c = cos(angle);
    double s = sin(angle);
    return glm::dvec2{a.x * c - a.y * s, a.x * s + a.y * c};
}

int main(int argc, char *argv[])
{

    // args -> resolution, incircle matrix?, eps, rotation

    glm::dvec2 a(-1.0, 0.0), b(0.0, -1.0), c(1.0, 0.0), d(0.0, 1.0);

    double rotation = 0.0;
    if (argc > 5)
    {
        rotation = atof(argv[5]);
    }

    a = rotate2d(a, H_PI * rotation);
    b = rotate2d(b, H_PI * rotation);
    c = rotate2d(c, H_PI * rotation);
    d = rotate2d(d, H_PI * rotation);

    double reasonable_eps = 2.0 * 1e-7; // default reasonable eps
    int exponent_x = 23;                // default exponent for 32 bit float

    if (argc > 3 && atoi(argv[3]) > 0)
    {
        // max -53
        printf("Using exponentx %d\n", atoi(argv[3]));
        exponent_x = atoi(argv[3]);
        // reasonable_eps = 1.0 / (1 << atoi(argv[3]));
        reasonable_eps = pow(2.0, -exponent_x);
    }
    printf("Exponent found: %d Perturbation used: %Lf\n", exponent_x, reasonable_eps);

    int resolution = 16384;
    if (argc > 1 && atoi(argv[1]) > 0)
    {
        resolution = atoi(argv[1]);
    }
    double *d_predicates, *h_predicates = new double[resolution * resolution];
    hipMalloc((void **)&d_predicates, resolution * resolution * sizeof(double));
    char *d_image, *h_image = new char[3 * resolution * resolution];
    hipMalloc((void **)&d_image, resolution * resolution * 3 * sizeof(char));

    const int blocksize = 128;
    dim3 dimBlock(blocksize);
    dim3 dimGrid((resolution * resolution + blocksize - 1) / dimBlock.x);

    if (argc > 2 && atoi(argv[2]) == 1)
    {
        printf("Writting matrix_incircle\n");
        calc_predicate_incircle<<<dimGrid, dimBlock>>>(resolution, a, b, c, d, reasonable_eps, d_predicates);
        hipDeviceSynchronize();
        get_image_from_incircle<<<dimGrid, dimBlock>>>(resolution, d_predicates, d_image);
        hipDeviceSynchronize();
        hipMemcpy(h_image, d_image, resolution * resolution * 3 * sizeof(char), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());
    }
    else if (argc > 2 && atoi(argv[2]) == 2)
    {
        printf("Writting exact_incircle\n");
        for (int i = 0; i < resolution * resolution; i++)
        {
            int j = i % resolution;
            int k = i / resolution;
            glm::dvec2 perturbation((k - resolution / 2) * reasonable_eps, (j - resolution / 2) * reasonable_eps);
            h_predicates[i] = exact_incircle(a, b, c, d + perturbation);
        }
        hipMemcpy(d_predicates, h_predicates, resolution * resolution * sizeof(double), hipMemcpyHostToDevice);
        get_image_from_incircle<<<dimGrid, dimBlock>>>(resolution, d_predicates, d_image);
        hipDeviceSynchronize();
        hipMemcpy(h_image, d_image, resolution * resolution * 3 * sizeof(char), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());
    }
    else
    {
        printf("Writting angle incircle\n");
        calc_predicate_angle_incircle<<<dimGrid, dimBlock>>>(resolution, a, b, c, d, reasonable_eps, d_predicates);
        hipDeviceSynchronize();
        get_image_from_angle_incircle<<<dimGrid, dimBlock>>>(resolution, d_predicates, d_image);
        hipDeviceSynchronize();
        hipMemcpy(h_image, d_image, resolution * resolution * 3 * sizeof(char), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        gpuErrchk(hipPeekAtLastError());
    }

    stbi_flip_vertically_on_write(true);

    if (argc > 4)
    {
        stbi_write_png(argv[4], resolution, resolution, 3, h_image, 0);
    }
    else
        stbi_write_png("res.bmp", resolution, resolution, 3, h_image, 0);
    return 0;
}